#include <hip/hip_runtime.h>

#include "cudaUtils.cuh"
extern "C" {
#include "ELLMatrix.h"
#include "Vector.h"
#include "SpMVResult.h"
#include "SpMV.h"
}

#include "SpMVKernel.cuh"

extern "C" void ELLMatrixHyb_SpMV_GPU(const ELLMatrix *ellMatrix, const COOMatrix *cooMatrix, const Vector *x, Vector *y, SpMVResultCUDA *result) {
    float *d_ellmatrix_data, *d_x, *d_y;
    u_int64_t *d_ellmatrix_col_index;
    hipEvent_t start, stop, instart, instop, outstart, outstop;
    size_t memoryUsed;
    hipDeviceProp_t prop;
    BlockGridInfo ellBlockGridInfo;
    SpMVResultCPU cooresult;
    if (!cooMatrix || !ellMatrix || !x || !y) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (x->size != cooMatrix->col_size && y->size != cooMatrix->row_size &&
        cooMatrix->row_size != ellMatrix->row_size && cooMatrix->col_size != cooMatrix->row_size) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (result) {
        memset(result, 0, sizeof(*result));
    }

    memoryUsed = (ellMatrix->data_size + x->size + y->size) * sizeof(float) + sizeof(u_int64_t) * (ellMatrix->data_size);
    int bestDev = CudaUtils_getBestDevice(memoryUsed);
    if (bestDev == -1) {
        fprintf(stderr,"%s\n", "NOT ENOUGH MEMORY");
        exit(EXIT_FAILURE);
    }
    CudaUtils_setDevice(bestDev);
    CudaUtils_getDeviceProp(bestDev, &prop);
    CudaUtils_getBestCudaParameters(ellMatrix->row_size, &prop, &ellBlockGridInfo);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&instart);
    hipEventCreate(&instop);
    hipEventCreate(&outstart);
    hipEventCreate(&outstop);

    hipEventRecord(instart);

    checkCudaErrors(hipMalloc(&(d_x), x->size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_y), y->size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_ellmatrix_data), ellMatrix->data_size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_ellmatrix_col_index), ellMatrix->data_size * sizeof (u_int64_t)));
    checkCudaErrors(hipMemcpy(d_x, x->data, x->size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, y->data, y->size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_ellmatrix_data, ellMatrix->data, ellMatrix->data_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_ellmatrix_col_index, ellMatrix->col_index, ellMatrix->num_non_zero_elements * sizeof(u_int64_t), hipMemcpyHostToDevice));
    hipEventRecord(instop);
    hipEventRecord(start);
    SpMV_ELL_kernel<<<ellBlockGridInfo.gridSize, ellBlockGridInfo.blockSize>>>(ellMatrix->row_size, d_ellmatrix_data, d_ellmatrix_col_index, ellMatrix->num_elem, d_x, d_y);
    hipEventRecord(stop);
    COOMatrix_SpMV_CPU(cooMatrix, x, y, &cooresult);
    hipEventRecord(outstart);
    checkCudaErrors(hipMemcpy(y->data, d_y, y->size * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_ellmatrix_data));
    checkCudaErrors(hipFree(d_ellmatrix_col_index));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    hipEventRecord(outstop);
    if (result) {
        result->success = 1;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&result->GPUKernelExecutionTime, start, stop);
        hipEventSynchronize(instop);
        hipEventElapsedTime(&result->GPUInputOnDeviceTime, instart, instop);
        hipEventSynchronize(outstop);
        hipEventElapsedTime(&result->GPUOutputFromDeviceTime, outstart, outstop);
        hipEventElapsedTime(&result->GPUTotalTime, instart, outstop);
        result->CPUTime = cooresult.timeElapsed;
        return;
    }
}

extern "C" void ELLMatrixHyb_SpMV_GPU_wpm(const ELLMatrix *ellMatrix, const COOMatrix *cooMatrix, const Vector *x, Vector *y, SpMVResultCUDA *result) {
    float *d_ellmatrix_data, *d_x, *d_y;
    u_int64_t *d_ellmatrix_col_index;
    hipEvent_t start, stop, instart, instop, outstart, outstop;
    size_t memoryUsed;
    hipDeviceProp_t prop;
    BlockGridInfo ellBlockGridInfo;
    SpMVResultCPU cooresult;
    if (!cooMatrix || !ellMatrix || !x || !y) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (x->size != cooMatrix->col_size && y->size != cooMatrix->row_size &&
        cooMatrix->row_size != ellMatrix->row_size && cooMatrix->col_size != cooMatrix->row_size) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (result) {
        memset(result, 0, sizeof(*result));
    }

    memoryUsed = (ellMatrix->data_size + x->size + y->size) * sizeof(float) + sizeof(u_int64_t) * (ellMatrix->data_size);
    int bestDev = CudaUtils_getBestDevice(memoryUsed);
    if (bestDev == -1) {
        fprintf(stderr,"%s\n", "NOT ENOUGH MEMORY");
        exit(EXIT_FAILURE);
    }
    CudaUtils_setDevice(bestDev);
    CudaUtils_getDeviceProp(bestDev, &prop);
    CudaUtils_getBestCudaParameters(ellMatrix->row_size, &prop, &ellBlockGridInfo);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&instart);
    hipEventCreate(&instop);
    hipEventCreate(&outstart);
    hipEventCreate(&outstop);

    hipEventRecord(instart);

    checkCudaErrors(hipMalloc(&(d_x), x->size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_y), y->size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_ellmatrix_data), ellMatrix->data_size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_ellmatrix_col_index), ellMatrix->data_size * sizeof (u_int64_t)));
    checkCudaErrors(hipMemcpyAsync(d_x, x->data, x->size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(d_y, y->data, y->size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(d_ellmatrix_data, ellMatrix->data, ellMatrix->data_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(d_ellmatrix_col_index, ellMatrix->col_index, ellMatrix->num_non_zero_elements * sizeof(u_int64_t), hipMemcpyHostToDevice));
    hipEventRecord(instop);
    hipEventRecord(start);
    SpMV_ELL_kernel<<<ellBlockGridInfo.gridSize, ellBlockGridInfo.blockSize>>>(ellMatrix->row_size, d_ellmatrix_data, d_ellmatrix_col_index, ellMatrix->num_elem, d_x, d_y);
    hipEventRecord(stop);
    COOMatrix_SpMV_CPU(cooMatrix, x, y, &cooresult);
    hipEventRecord(outstart);
    checkCudaErrors(hipMemcpy(y->data, d_y, y->size * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_ellmatrix_data));
    checkCudaErrors(hipFree(d_ellmatrix_col_index));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    hipEventRecord(outstop);
    if (result) {
        result->success = 1;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&result->GPUInputOnDeviceTime, start, stop);
        hipEventSynchronize(instop);
        hipEventElapsedTime(&result->GPUKernelExecutionTime, instart, instop);
        hipEventSynchronize(outstop);
        hipEventElapsedTime(&result->GPUOutputFromDeviceTime, outstart, outstop);
        hipEventElapsedTime(&result->GPUTotalTime, instart, outstop);
        result->CPUTime = cooresult.timeElapsed;
        return;
    }
}

