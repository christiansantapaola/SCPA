#include <hip/hip_runtime.h>

#include "cudaUtils.cuh"
extern "C" {
#include "ELLMatrix.h"
#include "Vector.h"
#include "SpMVResult.h"
#include "SpMV.h"
}

#include "SpMVKernel.cuh"

extern "C" void ELLMatrixHyb_SpMV_GPU(const ELLMatrix *ellMatrix, const COOMatrix *cooMatrix, const Vector *x, Vector *y, SpMVResultCUDA *result) {
    float *d_coomatrix_data, *d_ellmatrix_data, *d_x, *d_y;
    u_int64_t *d_coomatrix_col_index, *d_coomatrix_row_index, *d_ellmatrix_col_index;
    hipEvent_t start, stop, instart, instop, outstart, outstop;
    hipEvent_t cooStart, cooStop, cooInstart, cooInstop, cooOutstart, cooOutstop;
    size_t memoryUsed;
    hipDeviceProp_t prop;
    BlockGridInfo cooBlockGridInfo, ellBlockGridInfo;
    if (!cooMatrix || !ellMatrix || !x || !y) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (x->size != cooMatrix->col_size && y->size != cooMatrix->row_size &&
        cooMatrix->row_size != ellMatrix->row_size && cooMatrix->col_size != cooMatrix->row_size) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (result) {
        memset(result, 0, sizeof(*result));
    }

    memoryUsed = (ellMatrix->data_size + x->size + y->size) * sizeof(float) + sizeof(u_int64_t) * (ellMatrix->data_size);
    int bestDev = CudaUtils_getBestDevice(memoryUsed);
    if (bestDev == -1) {
        fprintf(stderr,"%s\n", "NOT ENOUGH MEMORY");
        exit(EXIT_FAILURE);
    }
    CudaUtils_setDevice(bestDev);
    CudaUtils_getDeviceProp(bestDev, &prop);
    CudaUtils_getBestCudaParameters(ellMatrix->row_size, &prop, &ellBlockGridInfo);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&instart);
    hipEventCreate(&instop);
    hipEventCreate(&outstart);
    hipEventCreate(&outstop);
    hipEventCreate(&cooStart);
    hipEventCreate(&cooStop);
    hipEventCreate(&cooInstart);
    hipEventCreate(&cooInstop);
    hipEventCreate(&cooOutstart);
    hipEventCreate(&cooOutstop);

    hipEventRecord(instart);

    checkCudaErrors(hipMalloc(&(d_x), x->size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_y), y->size * sizeof (float )));

    checkCudaErrors(hipMalloc(&(d_ellmatrix_data), ellMatrix->data_size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_ellmatrix_col_index), ellMatrix->data_size * sizeof (u_int64_t)));

    checkCudaErrors(hipMemcpy(d_x, x->data, x->size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, y->data, y->size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_ellmatrix_data, ellMatrix->data, ellMatrix->data_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_ellmatrix_col_index, ellMatrix->col_index, ellMatrix->num_non_zero_elements * sizeof(u_int64_t), hipMemcpyHostToDevice));

    hipEventRecord(instop);
    hipEventRecord(start);
    SpMV_ELL_kernel<<<ellBlockGridInfo.gridSize, ellBlockGridInfo.blockSize>>>(ellMatrix->row_size, d_ellmatrix_data, d_ellmatrix_col_index, ellMatrix->num_elem, d_x, d_y);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
    hipEventRecord(stop);
    checkCudaErrors(hipFree(d_ellmatrix_data));
    checkCudaErrors(hipFree(d_ellmatrix_col_index));

    memoryUsed = (cooMatrix->num_non_zero_elements + x->size + y->size) * sizeof(float) + sizeof(u_int64_t) * (2 * cooMatrix->num_non_zero_elements);
    bestDev = CudaUtils_getBestDevice(memoryUsed);
    if (bestDev == -1) {
        fprintf(stderr,"%s\n", "NOT ENOUGH MEMORY");
        exit(EXIT_FAILURE);
    }
    CudaUtils_setDevice(bestDev);
    CudaUtils_getDeviceProp(bestDev, &prop);
    CudaUtils_getBestCudaParameters(cooMatrix->num_non_zero_elements, &prop, &cooBlockGridInfo);

    hipEventRecord(cooInstart);

    checkCudaErrors(hipMalloc(&d_coomatrix_data, cooMatrix->num_non_zero_elements * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_coomatrix_col_index, cooMatrix->num_non_zero_elements * sizeof(u_int64_t)));
    checkCudaErrors(hipMalloc(&d_coomatrix_row_index, cooMatrix->num_non_zero_elements * sizeof(u_int64_t)));

    checkCudaErrors(hipMemcpy(d_coomatrix_data, cooMatrix->data, cooMatrix->num_non_zero_elements * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_coomatrix_col_index, cooMatrix->col_index, cooMatrix->num_non_zero_elements * sizeof(u_int64_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_coomatrix_row_index, cooMatrix->row_index, cooMatrix->num_non_zero_elements * sizeof(u_int64_t), hipMemcpyHostToDevice));

    hipEventRecord(cooInstop);
    hipEventRecord(cooStart);
    SpMV_COO_kernel<<<cooBlockGridInfo.gridSize, cooBlockGridInfo.blockSize>>>(cooMatrix->num_non_zero_elements,
                                                                               d_coomatrix_data,
                                                                               d_coomatrix_col_index,
                                                                               d_coomatrix_row_index,
                                                                               d_x,
                                                                               d_y);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
    hipEventRecord(cooStop);

    hipEventRecord(outstart);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipMemcpy(y->data, d_y, y->size * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_coomatrix_data));
    checkCudaErrors(hipFree(d_coomatrix_col_index));
    checkCudaErrors(hipFree(d_coomatrix_row_index));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    hipEventRecord(outstop);
    if (result) {
        float ellExTime, ellInTime, cooExTime, cooInTime;
        result->success = 1;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ellExTime, start, stop);
        hipEventSynchronize(instop);
        hipEventElapsedTime(&ellInTime, instart, instop);
        hipEventSynchronize(cooStop);
        hipEventElapsedTime(&cooExTime, cooStart, cooStop);
        hipEventSynchronize(cooInstop);
        hipEventElapsedTime(&cooInTime, cooInstart, cooInstop);
        hipEventSynchronize(outstop);
        hipEventElapsedTime(&result->GPUOutputFromDeviceTime, outstart, outstop);
        result->GPUKernelExecutionTime = ellExTime + cooExTime;
        result->GPUInputOnDeviceTime = ellInTime + cooInTime;
        result->GPUTotalTime = result->GPUInputOnDeviceTime + result->GPUKernelExecutionTime + result->GPUOutputFromDeviceTime;
        return;
    }
}

extern "C" void ELLMatrixHyb_SpMV_GPU_wpm(const ELLMatrix *ellMatrix, const COOMatrix *cooMatrix, const Vector *x, Vector *y, SpMVResultCUDA *result) {
    float *d_coomatrix_data, *d_ellmatrix_data, *d_x, *d_y;
    u_int64_t *d_coomatrix_col_index, *d_coomatrix_row_index, *d_ellmatrix_col_index;
    hipEvent_t start, stop, instart, instop, outstart, outstop;
    hipEvent_t cooStart, cooStop, cooInstart, cooInstop;
    size_t memoryUsed;
    hipDeviceProp_t prop;
    BlockGridInfo cooBlockGridInfo, ellBlockGridInfo;
    SpMVResultCPU cooresult;
    if (!cooMatrix || !ellMatrix || !x || !y) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (x->size != cooMatrix->col_size && y->size != cooMatrix->row_size &&
        cooMatrix->row_size != ellMatrix->row_size && cooMatrix->col_size != cooMatrix->row_size) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (result) {
        memset(result, 0, sizeof(*result));
    }

    memoryUsed = (ellMatrix->data_size + x->size + y->size) * sizeof(float) + sizeof(u_int64_t) * (ellMatrix->data_size);
    int bestDev = CudaUtils_getBestDevice(memoryUsed);
    if (bestDev == -1) {
        fprintf(stderr,"%s\n", "NOT ENOUGH MEMORY");
        exit(EXIT_FAILURE);
    }
    CudaUtils_setDevice(bestDev);
    CudaUtils_getDeviceProp(bestDev, &prop);
    CudaUtils_getBestCudaParameters(ellMatrix->row_size, &prop, &ellBlockGridInfo);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&instart);
    hipEventCreate(&instop);
    hipEventCreate(&outstart);
    hipEventCreate(&outstop);
    hipEventCreate(&cooStart);
    hipEventCreate(&cooStop);
    hipEventCreate(&cooInstart);
    hipEventCreate(&cooInstop);

    hipEventRecord(instart);

    checkCudaErrors(hipMalloc(&(d_x), x->size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_y), y->size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_ellmatrix_data), ellMatrix->data_size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_ellmatrix_col_index), ellMatrix->data_size * sizeof (u_int64_t)));
    checkCudaErrors(hipMemcpyAsync(d_x, x->data, x->size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(d_y, y->data, y->size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(d_ellmatrix_data, ellMatrix->data, ellMatrix->data_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(d_ellmatrix_col_index, ellMatrix->col_index, ellMatrix->num_non_zero_elements * sizeof(u_int64_t), hipMemcpyHostToDevice));
    hipEventRecord(instop);
    hipEventRecord(start);
    SpMV_ELL_kernel<<<ellBlockGridInfo.gridSize, ellBlockGridInfo.blockSize>>>(ellMatrix->row_size, d_ellmatrix_data, d_ellmatrix_col_index, ellMatrix->num_elem, d_x, d_y);
    hipEventRecord(stop);
    COOMatrix_SpMV_CPU(cooMatrix, x, y, &cooresult);
    hipEventRecord(outstart);
    checkCudaErrors(hipMemcpy(y->data, d_y, y->size * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_ellmatrix_data));
    checkCudaErrors(hipFree(d_ellmatrix_col_index));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    hipEventRecord(outstop);
    if (result) {
        float ellExTime, ellInTime, cooExTime, cooInTime;
        result->success = 1;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ellExTime, start, stop);
        hipEventSynchronize(instop);
        hipEventElapsedTime(&ellInTime, instart, instop);
        hipEventSynchronize(cooStop);
        hipEventElapsedTime(&cooExTime, cooStart, cooStop);
        hipEventSynchronize(cooInstop);
        hipEventElapsedTime(&cooInTime, cooInstart, cooInstop);
        hipEventSynchronize(outstop);
        hipEventElapsedTime(&result->GPUOutputFromDeviceTime, outstart, outstop);
        result->GPUKernelExecutionTime = ellExTime;
        result->GPUInputOnDeviceTime = ellInTime;
        hipEventElapsedTime(&result->GPUTotalTime, instart, outstop);
        result->CPUTime = cooresult.timeElapsed;
        return;
    }
}

