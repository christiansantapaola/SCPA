#include "hip/hip_runtime.h"
extern "C" {
#include "COOMatrix.h"
#include "Vector.h"
#include "SpMVResult.h"
}
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cudaUtils.cuh"



__global__ void
SpMV_COO_kernel(u_int64_t num_elements, const float *data, const u_int64_t *col_index, const u_int64_t *row_index, const float *x, float *y) {
    u_int64_t elem = blockIdx.x * blockDim.x + threadIdx.x;
    if (elem < num_elements) {
        atomicAdd(&y[row_index[elem]], data[elem] * x[col_index[elem]]);
    }
}

extern "C"
void COOMatrix_SpMV_GPU(const COOMatrix *matrix, const Vector *x, Vector *y, SpMVResultCUDA *result) {
    float *d_matrix_data, *d_x, *d_y;
    u_int64_t *d_col_index, *d_row_index;
    hipEvent_t start, stop, instart, instop, outstart, outstop;
    size_t memoryUsed;
    if (!matrix || !x || !y) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (x->size != matrix->col_size && y->size != matrix->row_size) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (result) {
        memset(result, 0, sizeof(*result));
    }
    memoryUsed = (matrix->num_non_zero_elements + x->size + y->size) * sizeof(float) +   sizeof(u_int64_t) * (2 * matrix->num_non_zero_elements);
    int bestDev = CudaUtils_getBestDevice(memoryUsed);
    if (bestDev == -1) {
        fprintf(stderr,"%s\n", "NOT ENOUGH MEMORY");
        exit(EXIT_FAILURE);
    }
    CudaUtils_setDevice(bestDev);
    hipDeviceProp_t prop;
    BlockGridInfo blockGridInfo;
    CudaUtils_getDeviceProp(bestDev, &prop);
    CudaUtils_getBestCudaParameters(matrix->num_non_zero_elements, &prop, &blockGridInfo);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&instart);
    hipEventCreate(&instop);
    hipEventCreate(&outstart);
    hipEventCreate(&outstop);


    hipEventRecord(instart);

    checkCudaErrors(hipMalloc(&d_matrix_data, matrix->num_non_zero_elements * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_col_index, matrix->num_non_zero_elements * sizeof(u_int64_t)));
    checkCudaErrors(hipMalloc(&d_row_index, matrix->num_non_zero_elements * sizeof(u_int64_t)));
    checkCudaErrors(hipMalloc(&d_x, matrix->row_size * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_y, matrix->row_size * sizeof(float)));

//    checkCudaErrors(hipMemcpyAsync(d_matrix_data, matrix->data,matrix->num_non_zero_elements * sizeof(float),hipMemcpyHostToDevice));
//    checkCudaErrors(hipMemcpyAsync(d_col_index, matrix->col_index, matrix->num_non_zero_elements * sizeof(u_int64_t), hipMemcpyHostToDevice));
//    checkCudaErrors(hipMemcpyAsync(d_row_index, matrix->row_index, (matrix->row_size + 1) * sizeof(u_int64_t), hipMemcpyHostToDevice));
//    checkCudaErrors(hipMemcpyAsync(d_x, x->data, matrix->row_size * sizeof(float), hipMemcpyHostToDevice));
//    checkCudaErrors(hipMemcpyAsync(d_y, y->data, matrix->row_size * sizeof(float), hipMemcpyHostToDevice));
//    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(d_matrix_data, matrix->data,matrix->num_non_zero_elements * sizeof(float),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_col_index, matrix->col_index, matrix->num_non_zero_elements * sizeof(u_int64_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_row_index, matrix->row_index, matrix->num_non_zero_elements * sizeof(u_int64_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, x->data, matrix->row_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, y->data, matrix->row_size * sizeof(float), hipMemcpyHostToDevice));

    hipEventRecord(instop);
    hipEventRecord(start);
    SpMV_COO_kernel<<<blockGridInfo.gridSize, blockGridInfo.blockSize>>>(matrix->num_non_zero_elements,
                                                                         d_matrix_data,
                                                                         d_col_index,
                                                                         d_row_index,
                                                                         d_x,
                                                                         d_y);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
    hipEventRecord(stop);

    hipEventRecord(outstart);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipMemcpy(y->data, d_y, matrix->row_size * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_matrix_data));
    checkCudaErrors(hipFree(d_col_index));
    checkCudaErrors(hipFree(d_row_index));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    hipEventRecord(outstop);
    hipEventSynchronize(stop);
    if (result) {
        result->success = 1;
        hipEventElapsedTime(&result->GPUKernelExecutionTime, start, stop);
        hipEventSynchronize(instop);
        hipEventElapsedTime(&result->GPUInputOnDeviceTime, instart, instop);
        hipEventSynchronize(outstop);
        hipEventElapsedTime(&result->GPUOutputFromDeviceTime, outstart, outstop);
        result->blockGridInfo = blockGridInfo;
        result->GPUusedGlobalMemory = memoryUsed;
        result->GPUtotalGlobMemory = prop.totalGlobalMem;
        return;
    }

}