#include "hip/hip_runtime.h"
extern "C" {
#include "CSRMatrix.h"
#include "Vector.h"
#include "SpMVResult.h"
#include "SpMV.h"
}
#include "cudaUtils.cuh"
#include <hip/hip_runtime.h>
#include "SpMVKernel.cuh"

#define MAX_X_SIZE 65536 / sizeof(float)

/*
 * Calcolo Performance:
 * Accessi alla memoria globale:
 * int row = blockIdx.x * blockDim.x + threadIdx.x; + 1
 * int row_start = row_ptr[row];                    + 1
 * int row_end = row_ptr[row + 1];                  + 1
 * data[elem]                                       + 1
 * x[col_index[elem]]                               + 2
 * y[row]                                           + 1
 * Totale                                           + 7
 * Numero Operazioni Float:
 * dot += data[elem] * x[col_index[elem]];          + 2
 * y[row] += dot;                                   + 1
 * Totale                                           + 3
 * Ratio float/access = 7/3
 */

__global__ void
SpMV_CSR_kernel(u_int64_t num_rows, const float *data, const u_int64_t *col_index, const u_int64_t *row_ptr, const float *x, float *y) {
    u_int64_t row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        float dot = 0.0f;
        u_int64_t row_start = row_ptr[row];
        u_int64_t row_end = row_ptr[row + 1];
        for (u_int64_t elem = row_start; elem < row_end; elem++) {
            dot += data[elem] * x[col_index[elem]];
        }
        y[row] += dot;
    }
}

extern "C"
int CSRMatrix_SpMV_CUDA(const CSRMatrix *d_matrix, const Vector *d_x, Vector *d_y, SpMVResultCUDA *result) {
    hipEvent_t start, stop;
    size_t memoryUsed;
    if (!d_matrix || !d_x || !d_y) {
        return SPMV_FAIL;
    }
    if (d_x->size != d_matrix->col_size && d_y->size != d_matrix->row_size) {
        return SPMV_FAIL;
    }
    memoryUsed = (d_matrix->num_non_zero_elements + d_x->size + d_y->size) * sizeof(float) + sizeof(u_int64_t) * (d_matrix->row_size + 1 + d_matrix->num_non_zero_elements);
    int bestDev = CudaUtils_getBestDevice(memoryUsed);
    if (bestDev == -1) {
        return SPMV_FAIL;
    }
    CudaUtils_setDevice(bestDev);
    hipDeviceProp_t prop;
    BlockGridInfo blockGridInfo;
    CudaUtils_getDeviceProp(bestDev, &prop);
    CudaUtils_getBestCudaParameters(d_matrix->row_size, &prop, &blockGridInfo);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    SpMV_CSR_kernel<<<blockGridInfo.gridSize, blockGridInfo.blockSize>>>(d_matrix->row_size,
                                                                         d_matrix->data,
                                                                         d_matrix->col_index,
                                                                         d_matrix->row_pointer,
                                                                         d_x->data,
                                                                         d_y->data);
    hipEventRecord(stop);

    if (result) {
        hipEventSynchronize(stop);
        hipEventElapsedTime(&result->GPUKernelExecutionTime, start, stop);
    }
    return SPMV_SUCCESS;
}