#include "hip/hip_runtime.h"
//
// Created by 9669c on 11/03/2022.
//


extern "C" {
#include "CSRMatrix.h"
#include "Vector.h"
#include "SpMVResult.h"
#include "SpMV.h"
}
#include "cudaUtils.cuh"
#include <hip/hip_runtime.h>

#define MAX_X_SIZE 65536 / sizeof(float)

/*
 * Calcolo Performance:
 * Accessi alla memoria globale:
 * int row = blockIdx.x * blockDim.x + threadIdx.x; + 1
 * int row_start = row_ptr[row];                    + 1
 * int row_end = row_ptr[row + 1];                  + 1
 * data[elem]                                       + 1
 * x[col_index[elem]]                               + 2
 * y[row]                                           + 1
 * Totale                                           + 7
 * Numero Operazioni Float:
 * dot += data[elem] * x[col_index[elem]];          + 2
 * y[row] += dot;                                   + 1
 * Totale                                           + 3
 * Ratio float/access = 7/3
 */

__global__ void
SpMV_CSR_kernel(int num_rows, const float *data, const int *col_index, const int *row_ptr, const float *x, float *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        float dot = 0.0f;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row + 1];
        for (int elem = row_start; elem < row_end; elem++) {
            dot += data[elem] * x[col_index[elem]];
        }
        y[row] += dot;
    }
}

extern "C"
void CSRMatrix_SpMV_GPU(const CSRMatrix *matrix,const Vector *x, Vector *y, SpMVResult *result) {
    float *d_matrix_data, *d_x, *d_y;
    int *d_col_index, *d_row_ptr;
    hipEvent_t start, stop, instart, instop, outstart, outstop;
    size_t memoryUsed;
    if (!matrix || !x || !y) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (x->size != matrix->col_size && y->size != matrix->row_size) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (result) {
        memset(result, 0, sizeof(*result));
    }
    memoryUsed = (matrix->num_non_zero_elements + x->size + y->size) * sizeof(float) +   sizeof(int) * (matrix->row_size + 1 + matrix->num_non_zero_elements);
    int bestDev = CudaUtils_getBestDevice();
    CudaUtils_setDevice(bestDev);
    hipDeviceProp_t prop;
    BlockGridInfo blockGridInfo;
    CudaUtils_getDeviceProp(bestDev, &prop);
    CudaUtils_getBestCudaParameters(matrix->row_size, &prop, &blockGridInfo);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&instart);
    hipEventCreate(&instop);
    hipEventCreate(&outstart);
    hipEventCreate(&outstop);


    hipEventRecord(instart);

    checkCudaErrors(hipMalloc(&d_matrix_data, matrix->num_non_zero_elements * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_col_index, matrix->num_non_zero_elements * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_row_ptr, (matrix->row_size + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_x, matrix->row_size * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_y, matrix->row_size * sizeof(float)));

    checkCudaErrors(hipMemcpyAsync(d_matrix_data, matrix->data,matrix->num_non_zero_elements * sizeof(float),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(d_col_index, matrix->col_index, matrix->num_non_zero_elements * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(d_row_ptr, matrix->row_pointer, (matrix->row_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(d_x, x->data, matrix->row_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyAsync(d_y, y->data, matrix->row_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());

    hipEventRecord(instop);
    hipEventRecord(start);
    SpMV_CSR_kernel<<<blockGridInfo.gridSize, blockGridInfo.blockSize>>>(matrix->row_size,
                                d_matrix_data,
                                d_col_index,
                                d_row_ptr,
                                d_x,
                                d_y);
    checkCudaErrors(hipDeviceSynchronize());
    hipEventRecord(stop);

    hipEventRecord(outstart);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipMemcpy(y->data, d_y, matrix->row_size * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_matrix_data));
    checkCudaErrors(hipFree(d_col_index));
    checkCudaErrors(hipFree(d_row_ptr));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    hipEventRecord(outstop);
    hipEventSynchronize(stop);
    if (result) {
        result->success = 1;
        hipEventElapsedTime(&result->GPUKernelExecutionTime, start, stop);
        hipEventSynchronize(instop);
        hipEventElapsedTime(&result->GPUInputOnDeviceTime, instart, instop);
        hipEventSynchronize(outstop);
        hipEventElapsedTime(&result->GPUOutputFromDeviceTime, outstart, outstop);
        result->blockGridInfo = blockGridInfo;
        result->GPUusedGlobalMemory = memoryUsed;
        result->GPUtotalGlobMemory = prop.totalGlobalMem;
        return;
    }

}