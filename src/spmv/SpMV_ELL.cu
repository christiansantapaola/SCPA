#include "hip/hip_runtime.h"
//
// Created by 9669c on 21/03/2022.
//

#include <hip/hip_runtime.h>

#include "cudaUtils.cuh"
#include "ELLMatrix.h"

__global__ void SpMV_ELL(int num_rows, const float *data, const int *col_index, int num_elem, const float *x, float *y) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < num_rows) {
        float dot = 0.0f;
        for (int i = 0; i < num_elem; i++) {
            int index = row + num_rows * i;
            dot += data[index] * x[col_index[index]];
        }
        y[row] += dot;
    }
}

SpMVResult ELLMatrix::SpMV_GPU(Vector &X, Vector &Y) {
    SpMVResult result = {false, 0, 0, 0,0};
    float *d_x, *d_y;
    float *d_data;
    int *d_col_index;
    hipEvent_t start, stop, instart, instop, outstart, outstop;
    if (X.getSize() != col_size && Y.getSize() != row_size) {
        result.success = false;
        return result;
    }
    CudaDeviceInfo deviceInfo = CudaDeviceInfo();
    deviceInfo.setDevice(deviceInfo.getBestDevice());
    size_t memory_used = X.getSize() * sizeof(float)  + Y.getSize() * sizeof(float) + data_size * sizeof(float) + data_size * sizeof(int);
    size_t memory_available = deviceInfo.getDeviceProp(deviceInfo.dev)->totalGlobalMem;
    printf("mem: %zu/%zu\n", memory_used, memory_available);
    if ( memory_used >= memory_available) {
        return result;
    }
    BlockGridInfo blockGridInfo = deviceInfo.getBlockSize(row_size);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&instart);
    hipEventCreate(&instop);
    hipEventCreate(&outstart);
    hipEventCreate(&outstop);

    auto t0 = std::chrono::high_resolution_clock::now();

    hipEventRecord(instart);
    checkCudaErrors(hipMalloc(&(d_x), X.getSize() * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_y), Y.getSize() * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_data), data_size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_col_index), data_size * sizeof (int )));

    checkCudaErrors(hipMemcpy(d_x, X.getData(), X.getSize() * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, Y.getData(), Y.getSize() * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_data, data, data_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_col_index, col_index, num_non_zero_elements * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(instop);
    hipEventRecord(start);
    SpMV_ELL<<<blockGridInfo.gridSize, blockGridInfo.blockSize>>>(row_size, d_data, d_col_index, num_elem, d_x, d_y);
    hipEventRecord(stop);
    hipEventRecord(outstart);
    checkCudaErrors(hipMemcpy(Y.getData(), d_y, Y.getSize() * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_col_index));
    hipEventRecord(outstop);
    auto t1 = std::chrono::high_resolution_clock::now();
    hipEventSynchronize(stop);
    hipEventElapsedTime(&result.GPUKernelExecutionTime, start, stop);
    hipEventSynchronize(instop);
    hipEventElapsedTime(&result.GPUInputOnDeviceTime, instart, instop);
    hipEventSynchronize(outstop);
    hipEventElapsedTime(&result.GPUOutputFromDeviceTime, outstart, outstop);
    result.CPUFunctionExecutionTime = std::chrono::duration_cast<std::chrono::nanoseconds>(t1 - t0).count();
    result.success = true;
    return result;
}