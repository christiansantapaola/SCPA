#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "cudaUtils.cuh"
extern "C" {
#include "ELLMatrix.h"
#include "Vector.h"
#include "SpMVResult.h"
#include "SpMV.h"
}

__global__ void SpMV_ELL(u_int64_t num_rows, const float *data, const u_int64_t *col_index, u_int64_t num_elem, const float *x, float *y) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < num_rows) {
        float dot = 0.0f;
        for (int i = 0; i < num_elem; i++) {
            int index = row + num_rows * i;
            dot += data[index] * x[col_index[index]];
        }
        y[row] += dot;
    }
}

void ELLMatrix_SpMV_GPU(const ELLMatrix *matrix,const Vector *x, Vector *y, SpMVResultCUDA *result) {
    float *d_x, *d_y;
    float *d_data;
    u_int64_t *d_col_index;
    hipEvent_t start, stop, instart, instop, outstart, outstop;
    size_t memoryUsed;
    if (!matrix || !x || !y) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (x->size != matrix->col_size && y->size != matrix->row_size) {
        if (result) {
            result->success = 0;
        }
        return;
    }
    if (result) {
        memset(result, 0, sizeof(*result));
    }
    memoryUsed = (matrix->data_size + x->size + y->size) * sizeof(float) +   sizeof(u_int64_t) * (matrix->data_size);
    int bestDev = CudaUtils_getBestDevice(memoryUsed);
    if (bestDev == -1) {
        fprintf(stderr,"%s\n", "NOT ENOUGH MEMORY");
        exit(EXIT_FAILURE);
    }
    CudaUtils_setDevice(bestDev);
    hipDeviceProp_t prop;
    BlockGridInfo blockGridInfo;
    CudaUtils_getDeviceProp(bestDev, &prop);
    CudaUtils_getBestCudaParameters(matrix->row_size, &prop, &blockGridInfo);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&instart);
    hipEventCreate(&instop);
    hipEventCreate(&outstart);
    hipEventCreate(&outstop);

    hipEventRecord(instart);
    checkCudaErrors(hipMalloc(&(d_x), x->size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_y), y->size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_data), matrix->data_size * sizeof (float )));
    checkCudaErrors(hipMalloc(&(d_col_index), matrix->data_size * sizeof (u_int64_t)));

//    checkCudaErrors(hipMemcpyAsync(d_x, x->data, x->size * sizeof(float), hipMemcpyHostToDevice));
//    checkCudaErrors(hipMemcpyAsync(d_y, y->data, y->size * sizeof(float), hipMemcpyHostToDevice));
//    checkCudaErrors(hipMemcpyAsync(d_data, matrix->data, matrix->data_size * sizeof(float), hipMemcpyHostToDevice));
//    checkCudaErrors(hipMemcpyAsync(d_col_index, matrix->col_index, matrix->num_non_zero_elements * sizeof(float), hipMemcpyHostToDevice));
//    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(d_x, x->data, x->size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, y->data, y->size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_data, matrix->data, matrix->data_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_col_index, matrix->col_index, matrix->num_non_zero_elements * sizeof(u_int64_t), hipMemcpyHostToDevice));
    hipEventRecord(instop);
    hipEventRecord(start);
    SpMV_ELL<<<blockGridInfo.gridSize, blockGridInfo.blockSize>>>(matrix->row_size, d_data, d_col_index, matrix->num_elem, d_x, d_y);
    checkCudaErrors(hipDeviceSynchronize());
    hipEventRecord(stop);
    checkCudaErrors(hipPeekAtLastError());
    hipEventRecord(outstart);
    checkCudaErrors(hipMemcpy(y->data, d_y, y->size * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_col_index));
    hipEventRecord(outstop);
    hipEventSynchronize(stop);
    if (result) {
        result->success = 1;
        hipEventElapsedTime(&result->GPUKernelExecutionTime, start, stop);
        hipEventSynchronize(instop);
        hipEventElapsedTime(&result->GPUInputOnDeviceTime, instart, instop);
        hipEventSynchronize(outstop);
        hipEventElapsedTime(&result->GPUOutputFromDeviceTime, outstart, outstop);
        result->blockGridInfo = blockGridInfo;
        result->GPUusedGlobalMemory = memoryUsed;
        result->GPUtotalGlobMemory = prop.totalGlobalMem;
        return;
    }
}