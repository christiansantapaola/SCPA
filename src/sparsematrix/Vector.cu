extern "C" {
#include "Vector.h"
}
#include <hip/hip_runtime.h>
#include "cudaUtils.cuh"

Vector *Vector_pinned_memory_new(unsigned int size) {
    Vector *vector = (Vector *) malloc(sizeof(Vector));
    vector->size = size;
    checkCudaErrors(hipHostAlloc(&vector->data, size * sizeof(float), hipHostMallocDefault));
    memset(vector->data, 0, size * sizeof(float));
    return vector;
}
void Vector_pinned_memory_free(Vector *vector) {
    if (!vector) return;
    checkCudaErrors(hipHostFree(vector->data));
    free(vector);
}
