extern "C" {
#include "Vector.h"
}
#include <hip/hip_runtime.h>
#include "cudaUtils.cuh"

Vector *Vector_new_wpm(unsigned int size) {
    Vector *vector = (Vector *) malloc(sizeof(Vector));
    vector->size = size;
    checkCudaErrors(hipHostAlloc(&vector->data, size * sizeof(float), hipHostMallocDefault));
    memset(vector->data, 0, size * sizeof(float));
    return vector;
}
void Vector_free_wpm(Vector *vector) {
    if (!vector) return;
    checkCudaErrors(hipHostFree(vector->data));
    free(vector);
}
