extern "C" {
#include "CSRMatrix.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cudaUtils.cuh"

extern "C" CSRMatrix *CSRMatrix_pinned_memory_new(COOMatrix *cooMatrix) {
    if (!cooMatrix) return NULL;
    CSRMatrix *csrMatrix = NULL;
    csrMatrix = (CSRMatrix *) malloc(sizeof(CSRMatrix));
    csrMatrix->row_size = cooMatrix->row_size;
    csrMatrix->col_size = cooMatrix->col_size;
    csrMatrix->num_non_zero_elements = cooMatrix->num_non_zero_elements;
    checkCudaErrors(hipHostAlloc(&csrMatrix->data, csrMatrix->num_non_zero_elements * sizeof(float ), hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc(&csrMatrix->col_index, csrMatrix->num_non_zero_elements * sizeof(u_int64_t), hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc(&csrMatrix->row_pointer, (csrMatrix->row_size + 1) * sizeof (u_int64_t), hipHostMallocDefault));
    Histogram *elemForRow = Histogram_new(csrMatrix->row_size);

    // mi calcolo prima la posizione in base alle righe, poi aggiungo il resto,
    // questo perchè gli elementi in COO non devono essere ordinati.
    for (u_int64_t i = 0; i < csrMatrix->num_non_zero_elements; i++) {
        Histogram_insert(elemForRow, cooMatrix->row_index[i]);
    }
    u_int64_t count = 0;
    for (u_int64_t i = 0; i < cooMatrix->row_size + 1; i++) {
        csrMatrix->row_pointer[i] = count;
        count += Histogram_getElemAtIndex(elemForRow, i);
    }
    /*
     * Qui uso un istogramma per salvarmi il numero di inserimenti alla riga i.
     */
    Histogram *elemInsertedForRow = Histogram_new(csrMatrix->row_size);
    for (u_int64_t i = 0; i < csrMatrix->num_non_zero_elements; i++) {
        u_int64_t row = cooMatrix->row_index[i];
        u_int64_t col = cooMatrix->col_index[i];
        float val = cooMatrix->data[i];
        int64_t offset = Histogram_getElemAtIndex(elemInsertedForRow, row);
        u_int64_t index = csrMatrix->row_pointer[row] + offset;
        csrMatrix->data[index] = val;
        csrMatrix->col_index[index] = col;
        Histogram_insert(elemInsertedForRow, row);
    }
    Histogram_free(elemForRow);
    Histogram_free(elemInsertedForRow);
    return csrMatrix;
}

void CSRMatrix_pinned_memory_free(CSRMatrix *csrMatrix) {
    if (!csrMatrix) return;
    checkCudaErrors(hipHostFree(csrMatrix->data));
    checkCudaErrors(hipHostFree(csrMatrix->col_index));
    checkCudaErrors(hipHostFree(csrMatrix->row_pointer));
    free(csrMatrix);
}