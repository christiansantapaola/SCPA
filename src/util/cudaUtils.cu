#include "cudaUtils.cuh"


void CudaUtils_setDevice(int device) {
    checkCudaErrors(hipSetDevice(device));
}
void CudaUtils_getDeviceProp(int device, hipDeviceProp_t *prop) {
    if (!prop) return;
    checkCudaErrors(hipGetDeviceProperties(prop, device));

}
int CudaUtils_getBestDevice(size_t memoryUsed) {
    int numDevices;
    hipDeviceProp_t *props;
    checkCudaErrors(hipGetDeviceCount(&numDevices));
    props = (hipDeviceProp_t *) malloc(sizeof(*props) * numDevices);
    for (int i = 0; i < numDevices; i++) {
        CudaUtils_getDeviceProp(i, &props[i]);
    }
    int bestDev = -1;
    int numSM = 0;
    int clockRate = 0;
    for (int i = 0; i < numDevices; i++) {
        if (memoryUsed > props[i].totalGlobalMem) {
            continue;
        }
        if (numSM < props[i].multiProcessorCount) {
            numSM = props[i].multiProcessorCount;
            bestDev = i;
            clockRate = props[i].clockRate;
        } else if ( numSM == props[i].multiProcessorCount ) {
            if (clockRate < props[i].clockRate) {
                numSM = props[i].multiProcessorCount;
                bestDev = i;
                clockRate = props[i].clockRate;
            }
        }
    }
    free(props);
    return bestDev;
}

int doesItFitInGlobalMemory(hipDeviceProp_t *prop, size_t size) {
    if (!prop) return 0;
    return size <= prop->totalGlobalMem;
}

void CudaUtils_getBestCudaParameters(u_int64_t numUnits, hipDeviceProp_t *prop, BlockGridInfo *bestParams) {
    if (!bestParams || !prop) return;
    u_int64_t size = 0;
    for (size = 1; prop->warpSize * size <= prop->maxThreadsPerBlock; size++);
    BlockGridInfo *infos = (BlockGridInfo *)malloc(size * sizeof(BlockGridInfo));
    memset(infos, 0, size * sizeof(BlockGridInfo));
    for (u_int64_t i = 1; prop->warpSize * i <= prop->maxThreadsPerBlock; i++) {
        infos[i - 1].maxThreadPerBlock = prop->maxThreadsPerBlock;
        infos[i - 1].maxBlockSizePerSM = prop->maxBlocksPerMultiProcessor;
        infos[i - 1].maxThreadPerSM = prop->maxThreadsPerMultiProcessor;
        infos[i - 1].blockSize =prop->warpSize * i;
        infos[i - 1].numBlockToFillSM = prop->maxThreadsPerMultiProcessor / infos[i - 1].blockSize;
        infos[i - 1].gridSize = (numUnits % infos[i - 1].blockSize == 0) ? numUnits / infos[i - 1].blockSize : numUnits / infos[i - 1].blockSize + 1;
        infos[i - 1].spread = (infos[i - 1].gridSize < (u_int64_t)prop->multiProcessorCount) ? (double) infos[i - 1].gridSize / (double) prop->multiProcessorCount : 1.0;
        infos[i - 1].utilizationSM = (double) infos[i - 1].gridSize / (double) infos[i - 1].numBlockToFillSM;
        infos[i - 1].numThread = infos[i - 1].blockSize * infos[i - 1].gridSize;
        infos[i - 1].wastedThread = infos[i - 1].numThread - numUnits;
        infos[i - 1].wastedThreadOverNumThread = (double) (infos[i - 1].wastedThread) / (double) infos[i - 1].numThread;
        infos[i - 1].utilization = infos[i - 1].utilizationSM + infos[i - 1].spread - infos[i - 1].wastedThreadOverNumThread;
    }
    int index = 0;
    double maxUtil = 0.0;
    for (int i = 0; i < size; i++) {
        if (infos[i].utilization > maxUtil) {
            maxUtil = infos[i].utilization;
            index = i;
        }
    }

    *bestParams = infos[index];
    free(infos);
}